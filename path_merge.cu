#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define X 0
#define Y 1
#define SIZEA 65536
#define SIZEB 65336

#define N_BLOCKS 64
#define N_THREADS 2

__global__ void mergeBig_k(int *A, int *B, int *M, int *A_idx, int *B_idx){

	// Shared memory on which we will work
	__shared__ int A_shared[1024]; // SIZEA / block size 
	__shared__ int B_shared[1024]; // SIZEB / block size

	__shared__ int biaisA;
	__shared__ int biaisB;

	// (endA-startA) : size of A in partition
	// (endB-startB) : size of B in partition
	int startA, endA;
	int startB, endB;
	
	// We retrieve the indexes of the start and end of A and B in relation to the global table
	if (blockIdx.x == 0){
		startA = 0;
		endA = A_idx[blockIdx.x];
		startB = 0;
		endB = B_idx[blockIdx.x];
	}
	else if (blockIdx.x == N_BLOCKS-1){
		startA = A_idx[blockIdx.x-1];
		endA = SIZEA;
		startB = B_idx[blockIdx.x-1];
		endB = SIZEB;
	}
	else{
		startA = A_idx[blockIdx.x-1];
		endA = A_idx[blockIdx.x];
		startB = B_idx[blockIdx.x-1];
		endB = B_idx[blockIdx.x];
	}

	// Article ratings
	// There are N elements to merge
	// N = SIZEA + SIZEB
	// Each partition contains N/p elements, each block processes a partition
	// N / p = (endB-startB) + (endA-startA) = (SIZEA+SIZEB) / N_BLOCKS
	// If Z is the number of threads
	// We are going to merge Z elements at a time
	// So we need to do it (N / p) / Z times
	// We are going to move the sliding window (N / p) / Z times
	int iter_max = (blockDim.x - 1 + (endB-startB) + (endA-startA)) / blockDim.x;
	int iter = 0;

	biaisA = 0;
	biaisB = 0;
	do{
		// To synchronize the biases
		__syncthreads();

		// Loading values ​​into shared memory
		if (startA + biaisA + threadIdx.x < endA){
			A_shared[threadIdx.x] = A[startA + biaisA + threadIdx.x];
		}

		if (startB + biaisB + threadIdx.x < endB){
			B_shared[threadIdx.x] = B[startB + biaisB + threadIdx.x];	
		}

		// To synchronize shared memory
		__syncthreads();

		// Get the size of the sliding window
		// In general it is the number of threads (blockDim.x), i.e. We are in a Z * Z square normally
		// But the size can be smaller if there are fewer blockDim.x elements to load
		int sizeAshared = endA-startA - biaisA;
		int sizeBshared = endB-startB - biaisB;
		if (sizeAshared < 0)
			sizeAshared = 0;
		if (sizeAshared > blockDim.x && sizeAshared != 0)
			sizeAshared = blockDim.x;
		if (sizeBshared < 0)
			sizeBshared = 0;
		if (sizeBshared > blockDim.x && sizeBshared != 0)
			sizeBshared = blockDim.x;

		// Binary search
		int i = threadIdx.x;

		if (i < sizeAshared + sizeBshared){
			int K[2];
			int P[2];

			if (i > sizeAshared) {
				K[X] = i - sizeAshared;
				K[Y] = sizeAshared;
				P[X] = sizeAshared;
				P[Y] = i - sizeAshared;
			}
			else {
				K[X] = 0;
				K[Y] = i;
				P[X] = i;
				P[Y] = 0;
			}

			while (1) {
				int offset = (abs(K[Y] - P[Y]))/2;
				int Q[2] = {K[X] + offset, K[Y] - offset};

				if (Q[Y] >= 0 && Q[X] <= sizeBshared && (Q[Y] == sizeAshared || Q[X] == 0 || A_shared[Q[Y]] > B_shared[Q[X]-1])) {
					if (Q[X] == sizeBshared || Q[Y] == 0 || A_shared[Q[Y]-1] <= B_shared[Q[X]]) {
						int idx = startA + startB + i + iter * blockDim.x;
						if (Q[Y] < sizeAshared && (Q[X] == sizeBshared || A_shared[Q[Y]] <= B_shared[Q[X]]) ) {
							M[idx] = A_shared[Q[Y]];
							atomicAdd(&biaisA, 1);	//  Bias to increment 
						}
						else {
							M[idx] = B_shared[Q[X]];
							atomicAdd(&biaisB, 1); //  Bias to increment
						}
						//printf("blockIdx.x = %d threadIdx.x = %d idx = %d m = %d biaisA = %d\n", blockIdx.x, threadIdx.x, idx, M[idx], biaisA);
						break ;
					}
					else {
						K[X] = Q[X] + 1;
						K[Y] = Q[Y] - 1;
					}
				}
				else {
					P[X] = Q[X] - 1;
					P[Y] = Q[Y] + 1 ;
				}
			}
		}
		iter = iter + 1;
	} while(iter < iter_max);
}

__global__ void pathBig_k(int *A, int *B, int *M, int *A_idx, int *B_idx){

	// In this kernel, we will simply look for diagonal N_BLOCKS
	// such that each block will process N/N_BLOCKS elements in the second kernel
	int i = (SIZEA + SIZEB)/N_BLOCKS * (blockIdx.x + 1);
	if (blockIdx.x == N_BLOCKS-1){
		return;
	}

	// Binary search
	int K[2];
	int P[2];

	if (i > SIZEA) {
		K[X] = i - SIZEA; //btop / abottom
		K[Y] = SIZEA; // atop 
		P[X] = SIZEA; // atop /bbottom
		P[Y] = i - SIZEA; //btop / abottom
	}
	else {
		K[X] = 0;
		K[Y] = i;
		P[X] = i;
		P[Y] = 0;
	}
	// abottom = btop
	while (1) {

		int offset = (abs(K[Y] - P[Y]))/2;  // (atop - abottom) / 2
		int Q[2] = {K[X] + offset, K[Y] - offset};

		if (Q[Y] >= 0 && Q[X] <= SIZEB && (Q[Y] == SIZEA || Q[X] == 0 || A[Q[Y]] > B[Q[X]-1])) {
			if (Q[X] == SIZEB || Q[Y] == 0 || A[Q[Y]-1] <= B[Q[X]]) {
				if (Q[Y] < SIZEA && (Q[X] == SIZEB || A[Q[Y]] <= B[Q[X]]) ) {
					M[i] = A[Q[Y]];
				}
				else {
					M[i] = B[Q[X]];
				}
				A_idx[blockIdx.x] = Q[Y];
				B_idx[blockIdx.x] = Q[X];
				// printf("blockIdx.x = %d | Aidx[%d] = %d | Bidx[%d] = %d \n", blockIdx.x, blockIdx.x, Q[Y], blockIdx.x, Q[X]);
				break ;
			}
			else {
				K[X] = Q[X] + 1;
				K[Y] = Q[Y] - 1;
			}
		}
		else {
			P[X] = Q[X] - 1;
			P[Y] = Q[Y] + 1;
		}
	}
}

int main(){

	// Allocation de la mémoire, remplissage du tableau
	int *A = (int*) malloc(sizeof(int) * SIZEA);
	for (int i = 0; i < SIZEA; i++){
		A[i] = 2 * i;
	}
	int *B = (int*) malloc(sizeof(int) * SIZEB);
	for (int i = 0; i < SIZEB; i++){
		B[i] = 2 * i + 1;
	}
	int mHost[SIZEA + SIZEB];		// Tableau merged	

	int A_idx[N_BLOCKS];			// Merge path
	int B_idx[N_BLOCKS];			// Merge path
	int *aDevice, *bDevice, *mDevice, *A_idxDevice, *B_idxDevice;

	// GPU global memory allocation
	hipMalloc( (void**) &aDevice, SIZEA * sizeof(int) );
	hipMalloc( (void**) &bDevice, SIZEB * sizeof(int) );
	hipMalloc( (void**) & , (SIZEA+SIZEB) * sizeof(int) );
	hipMalloc( (void**) &A_idxDevice, N_BLOCKS * sizeof(int) );
	hipMalloc( (void**) &B_idxDevice, N_BLOCKS * sizeof(int) );

	// Copy arrays to GPU
	hipMemcpy( aDevice, A, SIZEA * sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( bDevice, B, SIZEB * sizeof(int), hipMemcpyHostToDevice );

	// Run the kernel to find an array partition
	// (SIZEA+SIZEB) / N_BLOCKS elements to process for each block in the kernel
	pathBig_k<<<N_BLOCKS, 1>>>(aDevice, bDevice, mDevice, A_idxDevice, B_idxDevice);

//	hipMemcpy( mHost, mDevice, (SIZEA+SIZEB) * sizeof(int), hipMemcpyDeviceToHost );
//	hipMemcpy( A_idx, A_idxDevice, N_BLOCKS * sizeof(int), hipMemcpyDeviceToHost );
//	hipMemcpy( B_idx, B_idxDevice, N_BLOCKS * sizeof(int), hipMemcpyDeviceToHost );

//	A_idx[N_BLOCKS-1] = SIZEA;
//	B_idx[N_BLOCKS-1] = SIZEB;

//	hipMemcpy( A_idxDevice, A_idx, N_BLOCKS * sizeof(int), hipMemcpyHostToDevice );
//	hipMemcpy( B_idxDevice, B_idx, N_BLOCKS * sizeof(int), hipMemcpyHostToDevice );

	// Sliding window to load elements into shared memory
	mergeBig_k<<<N_BLOCKS, N_THREADS>>>(aDevice, bDevice, mDevice, A_idxDevice, B_idxDevice);

	hipMemcpy( mHost, mDevice, (SIZEA+SIZEB) * sizeof(int), hipMemcpyDeviceToHost );
	for (int i = 0; i < SIZEA+SIZEB; i ++){
		printf("m[%d] = %d\n", i, mHost[i]);
	}


	free(A);
	free(B);
	hipFree(aDevice);
	hipFree(bDevice);
	hipFree(mDevice);
	hipFree(A_idxDevice);
	hipFree(B_idxDevice);

	return 0;
}

